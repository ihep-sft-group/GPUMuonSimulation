#include "GPUSvc/FastMuon.h"
#include "GPUSvc/MuonSimu.cuh"
#include <iostream>

FastMuonMultiGPU::FastMuonMultiGPU(int gpuNum,std::string path,bool merge,std::vector<VoxelMethodHelper::StepInfo> &step_info):m_step_info(step_info)
{
    m_gpu_num = gpuNum;
    m_merge = merge;
    //加载数据
    m_data_pmt_x = NP<double>::Load((path + "pmt_x.npy").c_str());
    m_data_pmt_y = NP<double>::Load((path + "pmt_y.npy").c_str());
    m_data_pmt_z = NP<double>::Load((path + "pmt_z.npy").c_str());
    m_data_hit = NP<double>::Load((path + "hittime_cdf.npy").c_str());
    m_data_npe = NP<double>::Load((path + "npe_cdf.npy").c_str());
    //检查GPU
    std::cout<<"[CUDA init] version 03018 available GPU number: "<<gpulist.get_number_of_GPU()<<std::endl;
    assert(gpulist.get_number_of_GPU()>=m_gpu_num);
    gpulist.set_number_of_GPU(m_gpu_num);
    gpulist.search_gpu();
    if(!gpulist.check_p2p_just_gpu0()){
        std::cout<<"[CUDA ERROR] unable p2p copy"<<std::endl;
        // assert(gpulist.check_p2p_just_gpu0());
        // return;
        //不支持也能运行
    }
    gpulist.enable_p2p();
    
    for(auto& gpu:gpulist){
        dev_bufs_init_data[gpu.get_device_id()].clear();
        dev_bufs_hit_result_data[gpu.get_device_id()].clear();
        dev_bufs_npe_result_data[gpu.get_device_id()].clear();
        dev_bufs_step_data[gpu.get_device_id()].clear();
    }
}

void FastMuonMultiGPU::init()
{
    Extent ext = Extent(0);//不分块
    partitions = calc_partitions(gpulist, ext);//Partition < framework, data >
    scheduler.sync(partitions,[this](GPUPartition partition) { this->init(partition); });
}

void FastMuonMultiGPU::init(GPUPartition& partition)
{
    int gpu = partition.framework.gpu.get_device_id();
    CHECK(hipSetDevice(gpu));
    dev_bufs_init_data[gpu].push_back(create_and_alloc(m_data_pmt_x->data.size()));//pmt_x
    dev_bufs_init_data[gpu].push_back(create_and_alloc(m_data_pmt_y->data.size()));//pmt_y
    dev_bufs_init_data[gpu].push_back(create_and_alloc(m_data_pmt_z->data.size()));//pmt_z
    dev_bufs_init_data[gpu].push_back(create_and_alloc(m_data_hit->data.size()));
    dev_bufs_init_data[gpu].push_back(create_and_alloc(m_data_npe->data.size()));
    CHECK(hipMemcpy(dev_bufs_init_data[gpu][0]->get_ptr(),&(m_data_pmt_x->data[0]),dev_bufs_init_data[gpu][0]->get_size_in_bytes(),hipMemcpyHostToDevice));
    CHECK(hipMemcpy(dev_bufs_init_data[gpu][1]->get_ptr(),&(m_data_pmt_y->data[0]),dev_bufs_init_data[gpu][1]->get_size_in_bytes(),hipMemcpyHostToDevice));
    CHECK(hipMemcpy(dev_bufs_init_data[gpu][2]->get_ptr(),&(m_data_pmt_z->data[0]),dev_bufs_init_data[gpu][2]->get_size_in_bytes(),hipMemcpyHostToDevice));
    CHECK(hipMemcpy(dev_bufs_init_data[gpu][3]->get_ptr(),&(m_data_hit->data[0]),dev_bufs_init_data[gpu][3]->get_size_in_bytes(),hipMemcpyHostToDevice));
    CHECK(hipMemcpy(dev_bufs_init_data[gpu][4]->get_ptr(),&(m_data_npe->data[0]),dev_bufs_init_data[gpu][4]->get_size_in_bytes(),hipMemcpyHostToDevice));
    
    dev_bufs_hit_result_data[gpu].push_back(create_and_alloc(17746*3000));
    dev_bufs_npe_result_data[gpu].push_back(create_and_alloc(17746*3000));
}

void FastMuonMultiGPU::compute(std::vector<VoxelMethodHelper::StepInfo> &step_info,double*& result_hittime,double*& result_npe,int seed)
{
    m_step_info = step_info;
    m_seed = seed;
    Extent ext = Extent(m_step_info.size());
    // std::cout<<"m_step_info:"<<m_step_info.size()<<std::endl;
    partitions = calc_partitions(gpulist, ext);
    // std::cout<<"m_partition:"<<partitions.size()<<std::endl;
    //多GPU并行计算
    scheduler.sync(partitions,[this](GPUPartition partition) { this->compute(partition); });
    //合并结果
    reduce();
    //从GPU0返回结果
    CHECK(hipMemcpy(hittime_addr,dev_bufs_hit_result_data[0][0]->get_ptr(),dev_bufs_hit_result_data[0][0]->get_size_in_bytes(),hipMemcpyDeviceToHost));
    CHECK(hipMemcpy(hittime_npe,dev_bufs_npe_result_data[0][0]->get_ptr(),dev_bufs_npe_result_data[0][0]->get_size_in_bytes(),hipMemcpyDeviceToHost));
    result_hittime = hittime_addr;
    result_npe = hittime_npe;
}


void FastMuonMultiGPU::compute(const GPUPartition& partition)
{
    // GPU gpu = partition.framework.gpu;
    int gpu = partition.framework.gpu.get_device_id();
    // std::cout<<gpu<<" GPUid: "<<gpu<<std::endl;
    int data_length = partition.data.interval.second-partition.data.interval.first+1;
    CHECK(hipSetDevice(gpu));
    // Util::check("hipSetDevice [init]");
    // std::cout<<gpu<<" CHECK1: "<<data_length<<" "<<partition.data.interval.second<<" "<<partition.data.interval.first<<std::endl;
    //结果置0
    CHECK(hipMemset(dev_bufs_hit_result_data[gpu][0]->get_ptr(),0,dev_bufs_hit_result_data[gpu][0]->get_size_in_bytes()));
    CHECK(hipMemset(dev_bufs_npe_result_data[gpu][0]->get_ptr(),0,dev_bufs_npe_result_data[gpu][0]->get_size_in_bytes()));
    // std::cout<<gpu<<" CHECK2"<<std::endl;
    //转存step信息，占用大量栈内存，线程数量较少时容易耗尽栈内存,修改默认栈内存大小解决。ulimit -s 20480
    //使用堆内存可能产生线程安全问题，具体与编译器有关。
    // double v_data_step_r[data_length];
    // double v_data_step_pos_x[data_length];
    // double v_data_step_pos_y[data_length];
    // double v_data_step_pos_z[data_length];
    // double v_data_step_intPart[data_length];
    // double v_data_step_fraPart[data_length];
    // double v_data_step_starttime[data_length];
    //栈内存不够用，换成堆内存试试
    double* const v_data_step_r = new double[data_length];
    double* const v_data_step_pos_x = new double[data_length];
    double* const v_data_step_pos_y = new double[data_length];
    double* const v_data_step_pos_z = new double[data_length];
    double* const v_data_step_intPart = new double[data_length];
    double* const v_data_step_fraPart = new double[data_length];
    double* const v_data_step_starttime = new double[data_length];
    // std::cout<<gpu<<" CHECK3"<<std::endl;
    for (int i = 0; i < data_length; ++i)
    {
        // std::cout<<"[item]"<<i<<std::endl;
        v_data_step_r[i] = m_step_info[partition.data.interval.first+i].step_r;
        v_data_step_pos_x[i] = m_step_info[partition.data.interval.first+i].step_pos_x;
        v_data_step_pos_y[i] = m_step_info[partition.data.interval.first+i].step_pos_y;
        v_data_step_pos_z[i] = m_step_info[partition.data.interval.first+i].step_pos_z;
        v_data_step_intPart[i] = (double)m_step_info[partition.data.interval.first+i].step_energy_intPart;
        v_data_step_fraPart[i] = m_step_info[partition.data.interval.first+i].step_energy_fractionPart;
        v_data_step_starttime[i] = m_step_info[partition.data.interval.first+i].step_start_time;
    }
    // std::cout<<gpu<<" CHECK4"<<std::endl;
    //分配内存
    for(int i=0;i<7;++i){
        dev_bufs_step_data[gpu].push_back(create_and_alloc(data_length));
    }
    // std::cout<<gpu<<" CHECK5"<<std::endl;
    //拷贝step信息
    CHECK(hipMemcpy(dev_bufs_step_data[gpu][0]->get_ptr(),v_data_step_r,dev_bufs_step_data[gpu][0]->get_size_in_bytes(),hipMemcpyHostToDevice));
    CHECK(hipMemcpy(dev_bufs_step_data[gpu][1]->get_ptr(),v_data_step_pos_x,dev_bufs_step_data[gpu][1]->get_size_in_bytes(),hipMemcpyHostToDevice));
    CHECK(hipMemcpy(dev_bufs_step_data[gpu][2]->get_ptr(),v_data_step_pos_y,dev_bufs_step_data[gpu][2]->get_size_in_bytes(),hipMemcpyHostToDevice));
    CHECK(hipMemcpy(dev_bufs_step_data[gpu][3]->get_ptr(),v_data_step_pos_z,dev_bufs_step_data[gpu][3]->get_size_in_bytes(),hipMemcpyHostToDevice));
    CHECK(hipMemcpy(dev_bufs_step_data[gpu][4]->get_ptr(),v_data_step_intPart,dev_bufs_step_data[gpu][4]->get_size_in_bytes(),hipMemcpyHostToDevice));
    CHECK(hipMemcpy(dev_bufs_step_data[gpu][5]->get_ptr(),v_data_step_fraPart,dev_bufs_step_data[gpu][5]->get_size_in_bytes(),hipMemcpyHostToDevice));
    CHECK(hipMemcpy(dev_bufs_step_data[gpu][6]->get_ptr(),v_data_step_starttime,dev_bufs_step_data[gpu][6]->get_size_in_bytes(),hipMemcpyHostToDevice));
    // std::cout<<gpu<<" CHECK6"<<std::endl;
    //计算
    GPUExecConfig cfg;
    MuonSimu(cfg,dev_bufs_init_data[gpu],dev_bufs_step_data[gpu],dev_bufs_hit_result_data[gpu],dev_bufs_npe_result_data[gpu],m_seed+gpu,data_length);
    CHECK(hipDeviceSynchronize());
    // std::cout<<gpu<<" CHECK7"<<std::endl;
    dev_bufs_step_data[gpu].clear();
    delete []v_data_step_r;
    delete []v_data_step_pos_x;
    delete []v_data_step_pos_y;
    delete []v_data_step_pos_z;
    delete []v_data_step_intPart;
    delete []v_data_step_fraPart;
    delete []v_data_step_starttime;
}

void FastMuonMultiGPU::MuonSimu(GPUExecConfig& cfg,device_bufs_double& init_data,device_bufs_double& step_data, \
    device_bufs_double& hit_result, device_bufs_double& npe_result, int seed,int step_size)
{
    dim3 grid = cfg.get_grid();
    dim3 block = cfg.get_block();
    step_calculate_every_pmt<<<grid, block>>>(step_data[0]->get_ptr(),step_data[1]->get_ptr(),step_data[2]->get_ptr(), \
    step_data[3]->get_ptr(),init_data[0]->get_ptr(),init_data[1]->get_ptr(), \
    init_data[2]->get_ptr(),step_data[4]->get_ptr(),step_data[5]->get_ptr(), \
    step_data[6]->get_ptr(),17746,init_data[3]->get_ptr(),init_data[4]->get_ptr(),seed, \
    hit_result[0]->get_ptr(),npe_result[0]->get_ptr(),step_size,true); 
}

void FastMuonMultiGPU::reduce()
{
    if(gpulist.get_number_of_GPU()==1){
        return;
    }
    CHECK(hipSetDevice(0));
    // Util::check("hipSetDevice [init]");
    //拷贝结果到GPU0
    dev_bufs_hit_result_data[0].push_back(create_and_alloc(17746*3000));
    dev_bufs_npe_result_data[0].push_back(create_and_alloc(17746*3000));
    for(int gpuid = 1; gpuid < gpulist.get_number_of_GPU(); ++gpuid){
        CHECK(hipMemcpyPeer(dev_bufs_hit_result_data[0][1]->get_ptr(),0,dev_bufs_hit_result_data[gpuid][0]->get_ptr(),gpuid,dev_bufs_hit_result_data[0][1]->get_size_in_bytes()));
        CHECK(hipMemcpyPeer(dev_bufs_npe_result_data[0][1]->get_ptr(),0,dev_bufs_npe_result_data[gpuid][0]->get_ptr(),gpuid,dev_bufs_hit_result_data[0][1]->get_size_in_bytes()));
        dim3 block(1024);
        dim3 grid(51991);
        add<<<grid, block>>>(dev_bufs_hit_result_data[0][0]->get_ptr(), \
        dev_bufs_npe_result_data[0][0]->get_ptr(), dev_bufs_hit_result_data[0][1]->get_ptr(), \
        dev_bufs_npe_result_data[0][1]->get_ptr());
        CHECK(hipDeviceSynchronize());
    }
    //释放内存
    // dev_bufs_hit_result_data[0][1]->free();//无需手动释放
    // dev_bufs_npe_result_data[0][1]->free();
    dev_bufs_hit_result_data[0].pop_back();
    dev_bufs_npe_result_data[0].pop_back();
}


FastMuonMultiGPU::device_buf_double FastMuonMultiGPU::create_and_alloc(int size)
{
    Extent ext(size); 
    FastMuonMultiGPU::device_buf_double d = std::move(make_unique<DeviceBuffer<double>>(ext)); 
    d->alloc();
	return d;
}

FastMuonMultiGPU::device_buf_double FastMuonMultiGPU::create_and_alloc(Extent& ext)
{
    FastMuonMultiGPU::device_buf_double d = std::move(make_unique<DeviceBuffer<double>>(ext));
	d->alloc();
	return d;
}

void FastMuonMultiGPU::exit()
{
    for(auto& gpu:gpulist){
        dev_bufs_init_data[gpu.get_device_id()].clear();
        dev_bufs_hit_result_data[gpu.get_device_id()].clear();
        dev_bufs_npe_result_data[gpu.get_device_id()].clear();
        dev_bufs_step_data[gpu.get_device_id()].clear();
    }//先释放智能指针，再重置GPU
    Util::reset_all();
    return;
}