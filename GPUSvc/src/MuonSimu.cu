#include "hip/hip_runtime.h"
#include "GPUSvc/MuonSimu.cuh"

__global__ void
add(double* hit_res,double* npe_res,double* hit,double* npe)
{
    int id = blockIdx.x*blockDim.x+threadIdx.x;
    if (id < 53238000){//17746*3000
        if(hit_res[id]==0){
            hit_res[id] = hit[id];
        }
        else if(hit[id]!=0){
            hit_res[id] = (hit_res[id]<hit[id]) ? hit_res[id] : hit[id]; 
        }
        npe_res[id] += npe[id];
    }
}

__global__ void
step_calculate_every_pmt(double *r,double *pos_x,double *pos_y,double *pos_z,double *pmt_x,\
    double *pmt_y,double *pmt_z,double *intPart,double *fractionPart,double *start_time, \
    int numElements,double *hittime_histo,double *npe,int seed,double *hittime_result, \
    double *npe_result,int step_size,bool m_merge)
{
    int id = blockIdx.x*blockDim.x+threadIdx.x;
    //numElements = pmt numbers
    if (id < numElements){
        hiprandState state;
        generateRandomInit(&state,seed+id);
        
        Res_Arr pmt_arr;
        init_res_arr(&pmt_arr,hittime_result,npe_result,id);
        
        for(int i=0;i<step_size;i++)
        {
            calculate_by_step(r[i],pos_x[i],pos_y[i],pos_z[i],pmt_x[id],pmt_y[id],\
                pmt_z[id],intPart[i],fractionPart[i],start_time[i],hittime_histo,\
                npe,&state,&pmt_arr,m_merge);
        }
    }
}

__device__ void
calculate_by_step(double r,double pos_x,double pos_y,double pos_z,double pmt_x,double pmt_y,\
    double pmt_z,double intPart,double fractionPart,double start_time,double *hittime_histo,\
    double *npe,hiprandState *state,Res_Arr *p_pmt_arr,bool m_merge)
{
    double theta = calculateAngle(pmt_x,pmt_y,pmt_z,pos_x,pos_y,pos_z);
    for(int j = 0;j<intPart; ++j){
        generateHits(r,theta,1,start_time,hittime_histo,npe,state,p_pmt_arr,m_merge);
    }
    generateHits(r,theta,fractionPart,start_time,hittime_histo,npe,state,p_pmt_arr,m_merge);
}

__device__ double
calculateAngle(double x,double y,double z,double a,double b,double c)
{
    //  printf("x=%lf,y=%lf,z=%lf,a=%lf,b=%lf,c=%lf\n",x,y,z,a,b,c);
    double result = 0;
    if (a == 0 and b == 0 and c == 0){
        return result;
    }
    else{
        result = acos((a*x+b*y+c*z)/(norm3d(x,y,z)*norm3d(a,b,c)));
        //printf("result theta = %lf",result);
        return result;
    }
}

__device__ void 
generateHits(double r,double theta, double ratio,double start_time,double *hittime_histo,\
    double *npe,hiprandState *state,Res_Arr *p_r_arr,bool m_merge)
{
    int npe_histo_id = get_npe(r,theta,npe,state);
    if (npe_histo_id>0)
    {
        // int merge_index;
        // double hittime_single;
        for (int hitj = 0; hitj < npe_histo_id; ++hitj) 
        {
            // skip the photon according to the energy deposit
            if (ratio<1 and generateRandom(state)>ratio)
            {
                continue;
            }
            double hittime_single = start_time;
            hittime_single += (double)get_hittime(r, theta, 0, hittime_histo, state);
            // merge_index = get_hittime(r, theta, 0, hittime_histo, state);
            // hittime_single = start_time+merge_index;
            if(m_merge){
                // result_merge_save(p_r_arr,merge_index);
                result_merge_save(p_r_arr,hittime_single,(int)(hittime_single/1));
            }
            else{
                append_res_arr(p_r_arr,hittime_single);
            }
            
        }
    }
}

__device__ int
get_hittime(double r, double theta, int mode, double *hittime_histo, hiprandState *state) {
    int binx = get_bin_x(r);
    int biny = get_bin_y(theta);
    return get_hittime_bin(binx, biny, mode, hittime_histo, state);
}

__device__ int 
get_hittime_bin(int binx, int biny, int mode, double *hittime_histo, hiprandState *state) {
    // hit time = tmean + tres
    int hittime_single = 0;
    if (mode == 0) {
        hittime_single = get_hittime_all(binx,biny,hittime_histo,state);
    }
    return hittime_single;
}

__device__ int 
get_hittime_all(int binx, int biny,double *hittime_histo, hiprandState *state) {
    // TH1F* h = get_hist(binx, biny);
    const int xbinnum = 200;
    const int ybinnum = 180;
    if (binx<1) { binx = 1; }
    else if (binx > xbinnum) { binx = xbinnum;}
    if (biny<1) { biny = 1; }
    else if (biny > ybinnum) { biny = ybinnum;}
    int idx = (binx-1)*ybinnum+(biny-1);
    int hittime_single = sampling(state,hittime_histo,3000,idx);
    return hittime_single;
}


__device__ int 
get_bin_x(double r) 
{
    int binx = 1;
    int xmode = 2;
    if (xmode == 2) //KR
    {
        binx = r_findBin(r);
    } 
    return binx;
}

__device__ int 
get_bin_y(double theta) {
    int biny = 1;
    int ymode = 4;
    if (ymode == 4) {
        biny = theta_findBin(theta);
    }
    return biny;
}

__device__ int 
r_findBin(double r)
{
    const int binnum = 200;
    const double begin = 0;
    const double end = 17.7;
    if(r==0){
        return 1;
    }
    else{
        return (int)ceil((r-begin)/(end-begin)*binnum); 
    }
}

__device__ int 
get_npe(double r,double theta,double *npe,hiprandState *state)
{

    int binx = r3_findBin(pow(r,3));
    int biny = theta_findBin(theta);
    // printf("[npe] r=%lf,theta=%lf,binx=%d,biny=%d\n",r,theta,binx,biny);
    return get_npe_num(binx,biny,npe,state);
}

__device__ int 
r3_findBin(double r3)
{
    const int binnum = 100;
    const double begin = 0;
    const double end = 5600;
    if(r3 == 0){
        return 1;
    }
    else{
        return (int)ceil((r3-begin)/(end-begin)*binnum);
    }
}


__device__ int 
theta_findBin(double theta)
{
    const int binnum = 180;
    const double begin = 0; 
    const double end = 180.01*HIP_PI_F/180.0;
    if(theta == 0){
        
        return 1;
    }
    else{
        return (int)ceil((theta-begin)/(end-begin)*binnum);
    }
}

__device__ int 
get_npe_num(int binx,int biny,double *npe,hiprandState *state)
{
    int npe_from_single = 0;
    if (1 <= binx and binx <= 100 and 1 <= biny and biny <= 180) {
        npe_from_single = sampling(state,npe,33,(binx-1)*180+(biny-1));	
    } else if (binx==1 and (biny<1 or biny>180)) {
        biny = generateRandomInt(state,1,180);
        npe_from_single = sampling(state,npe,33,(binx-1)*180+(biny-1));	
    } else if (binx>1 and (biny<1 or biny>180)) {
        if (biny>180) { biny = 180; }
        else if (biny<1){ biny = 1; }
        npe_from_single = sampling(state,npe,33,(binx-1)*180+(biny-1));
    } else {
        static long warning = 0;
        ++warning;
        if (warning < 2) {
            // printf("npe lost: %d/%d\n", binx,biny);
        } else if (warning == 10) {
            // printf("too many npe lost complains.\n");
        }
    }
    return npe_from_single;
}

__device__ double
generateRandom(hiprandState *state)
{
    // int id = blockIdx.x*blockDim.x+threadIdx.x;   
    double result = abs(hiprand_uniform_double(state));
    return result;
}

__device__ int
generateRandomInt(hiprandState *state,int begin,int end)
{
    int result = begin+int(ceil(abs(hiprand_uniform_double(state))*(end-begin)));
    return result;
}

__device__ void
generateRandomInit(hiprandState *state,int seed)
{
    // printf("seed = %d\n",seed);
    int id = blockIdx.x*blockDim.x+threadIdx.x;   
    hiprand_init(seed, id, 0, state);
}

__device__ int 
sampling(hiprandState *state,double *histo,int max,int id)
{
    double prob; 
    prob = generateRandom(state);
    return binarySearch(histo,prob,max,id);
}

__device__ int
binarySearch(double *histo,double target,int max,int id)
{
    int start = 0;
    int end = max-1;
    int mid;
    while(start+1<end){
        mid = start+(end-start)/2;
        if (histo[id*max+mid]==target){
            end = mid;
        } 
        else if (histo[id*max+mid] < target){
            start = mid;
        }
        else if (histo[id*max+mid] > target){
            end = mid;
        }
    }
    if (target <= histo[id*max+start]){
        return start;
    }
    else if (histo[id*max+start] < target){
        return end;
    }
    
    return -1;
}

__device__ void
append_res_arr(Res_Arr *p, double val)//追加
{
    if(p->index==p->max-1){
        // printf("error! out of memory for result!\n");
        return;
        // assert(p->index<p->max);
    }
    p->hittime_arr[p->index+p->begin] = val;
    // p->pmtid_arr[p->index+p->begin] = p->id;
    p->npe_arr[p->id] += 1;
    p->index += 1;
    return;
}

// __device__ void 
// result_merge_save(Res_Arr *p,int val)
// {
//     p->hittime_arr[val+p->begin] +=1;
//     return;
// }

__device__ void 
result_merge_save(Res_Arr *p,double hittime,int index)
{
    if(index>3000){
        p->npe_arr[2999+p->begin] +=1;
        if(p->hittime_arr[2999+p->begin]==0 || p->hittime_arr[2999+p->begin]>hittime){
            p->hittime_arr[2999+p->begin] = hittime;
        }
        return;
    }
    p->npe_arr[index+p->begin] +=1;
    if(p->hittime_arr[index+p->begin]==0 || p->hittime_arr[index+p->begin]>hittime){
        p->hittime_arr[index+p->begin] = hittime;
    }
    return;
}

__device__ void
init_res_arr(Res_Arr *p,double *hittime_result,double *npe_result,int pmtid){
    p->hittime_arr = hittime_result;//存储的内存空间
    // p->pmtid_arr = pmt_res_arr;//存储hit对应的pmtid
    p->npe_arr = npe_result;
    p->index = 0;
    p->begin = pmtid*pmt_mem;//存储该pmt在数组中的起始存取点
    p->id = pmtid;
    p->max = pmt_mem;
    return;
}

inline void __cudaCheckError( const char *file, const int line )
{
#ifdef CUDA_ERROR_CHECK
    hipError_t err = hipGetLastError();
    if ( hipSuccess != err )
    {
        fprintf( stderr, "cudaCheckError() failed at %s:%i : %s\n",
                 file, line, hipGetErrorString( err ) );
        exit( -1 );
    }
    // Catch errors from CUDA kernel calls
    // More careful checking. However, this will affect performance.
    // Comment away if needed.
    // err = hipDeviceSynchronize();
    // if( hipSuccess != err )
    // {
    //     fprintf( stderr, "cudaCheckError() with sync failed at %s:%i : %s\n",
    //              file, line, hipGetErrorString( err ) );
    //     exit( -1 );
    // }
#endif

    return;
}

// void GPU_init_data(double *pmt_x,double *pmt_y,double *pmt_z,double *data_hit,double *data_npe,int size_1,int size_2,int size_3)
// {
//     CHECK(hipMalloc((double**)&g_pmt_x,size_1*sizeof(double)));
//     CHECK(hipMalloc((double**)&g_pmt_y,size_1*sizeof(double)));
//     CHECK(hipMalloc((double**)&g_pmt_z,size_1*sizeof(double)));
//     CHECK(hipMalloc((double**)&g_data_hit,size_2*sizeof(double)));
//     CHECK(hipMalloc((double**)&g_data_npe,size_3*sizeof(double)));
//     CHECK(hipMemcpy(g_pmt_x, pmt_x, size_1*sizeof(double), hipMemcpyHostToDevice));
//     CHECK(hipMemcpy(g_pmt_y, pmt_y, size_1*sizeof(double), hipMemcpyHostToDevice));
//     CHECK(hipMemcpy(g_pmt_z, pmt_z, size_1*sizeof(double), hipMemcpyHostToDevice));
//     CHECK(hipMemcpy(g_data_hit, data_hit, size_2*sizeof(double), hipMemcpyHostToDevice));
//     CHECK(hipMemcpy(g_data_npe, data_npe, size_3*sizeof(double), hipMemcpyHostToDevice));
//     //结果数组初始长度为3000，内存占用约1.3GB
//     CHECK(hipMalloc((double**)&d_hittime_result,pmt_num*pmt_mem*sizeof(double)));
//     CHECK(hipMalloc((int**)&d_result_size,pmt_num*sizeof(int)));
//     CHECK(hipMalloc((double**)&d_r,sizeof(double)));
//     CHECK(hipMalloc((double**)&d_pos_x,sizeof(double)));
//     CHECK(hipMalloc((double**)&d_pos_y,sizeof(double)));
//     CHECK(hipMalloc((double**)&d_pos_z,sizeof(double)));
//     CHECK(hipMalloc((double**)&d_intPart,sizeof(double)));
//     CHECK(hipMalloc((double**)&d_fractionPart,sizeof(double)));
//     CHECK(hipMalloc((double**)&d_start_time,sizeof(double)));
//     //设置内存
//     // CHECK(hipMemset(d_hittime_result,0,pmt_num*pmt_mem*sizeof(double)));
//     // CHECK(hipMemset(d_pmtid_result,0,pmt_num*pmt_mem*sizeof(int)));
// }

// void GPU_init_data_multi(double *pmt_x,double *pmt_y,double *pmt_z,double *data_hit,double *data_npe,int size_1,int size_2,int size_3,int gpunum)
// {
    
//     // int deviceCount = 0;
//     // CHECK(hipGetDeviceCount(&deviceCount));
//     // m_gpuNum = (gpunum<=deviceCount) ? gpunnum : deviceCount;
//     // assert(m_gpuNum!=0);
//     //GPU0
//     CHECK(hipSetDevice(0));
//     CHECK(hipMalloc((double**)&g_pmt_x,size_1*sizeof(double)));
//     CHECK(hipMalloc((double**)&g_pmt_y,size_1*sizeof(double)));
//     CHECK(hipMalloc((double**)&g_pmt_z,size_1*sizeof(double)));
//     CHECK(hipMalloc((double**)&g_data_hit,size_2*sizeof(double)));
//     CHECK(hipMalloc((double**)&g_data_npe,size_3*sizeof(double)));
//     CHECK(hipMemcpy(g_pmt_x, pmt_x, size_1*sizeof(double), hipMemcpyHostToDevice));
//     CHECK(hipMemcpy(g_pmt_y, pmt_y, size_1*sizeof(double), hipMemcpyHostToDevice));
//     CHECK(hipMemcpy(g_pmt_z, pmt_z, size_1*sizeof(double), hipMemcpyHostToDevice));
//     CHECK(hipMemcpy(g_data_hit, data_hit, size_2*sizeof(double), hipMemcpyHostToDevice));
//     CHECK(hipMemcpy(g_data_npe, data_npe, size_3*sizeof(double), hipMemcpyHostToDevice));
//     //结果数组初始长度为3000，内存占用约2.6GB
//     CHECK(hipMalloc((double**)&d_hittime_result,pmt_num*pmt_mem*sizeof(double)));
//     CHECK(hipMalloc((int**)&d_npe_result,pmt_num*pmt_mem*sizeof(int)));
//     //GPU 1-8
//     //连通性测试
//     // int canAccessPerr = 0;
//     // for(int i =0; i<m_gpuNum; ++i){
//     //     CHECK(hipDeviceCanAccessPeer(&canAccessPeer, i, -1));
//     //     hipDeviceCanAccessPeer(&canAccessPeer, i, m_gpuNum);
//     //     hipSetDevice(i);
//     //     hipDeviceEnablePeerAccess(i, 0);
//     // }

// }

// float GPU_Sampling_wrapper(double *r,double *pos_x,double *pos_y,double *pos_z, \
//     double *intPart, double *fractionPart,double *start_time,\
//     int seed,int step_size,double *hittime_addr,int *result_size)
// {
//     //debug
//     // size_t psize = 0;
//     // hipDeviceGetLimit(&psize,hipLimitPrintfFifoSize);
    
//     //GPU计时，设置开始和结束事件
//     hipEvent_t start, stop;
//     hipEvent_t gpu_start,gpu_stop,data_start,data_stop;
//     CHECK(hipEventCreate(&start));
//     CHECK(hipEventCreate(&stop));
//     CHECK(hipEventCreate(&gpu_start));
//     CHECK(hipEventCreate(&gpu_stop));
//     CHECK(hipEventCreate(&data_start));
//     CHECK(hipEventCreate(&data_stop));
//     CHECK(hipEventRecord(start));
//     CHECK(hipEventRecord(data_start));
    
//     CHECK(hipMemset(d_result_size,0,pmt_num*sizeof(int)));
//     //将CPU内存拷贝到GPU
//     CHECK(hipMemcpy(d_r, r, sizeof(double), hipMemcpyHostToDevice));
//     CHECK(hipMemcpy(d_pos_x, pos_x, sizeof(double), hipMemcpyHostToDevice));
//     CHECK(hipMemcpy(d_pos_y, pos_y, sizeof(double), hipMemcpyHostToDevice));
//     CHECK(hipMemcpy(d_pos_z, pos_z, sizeof(double), hipMemcpyHostToDevice));
//     CHECK(hipMemcpy(d_intPart, intPart, sizeof(double), hipMemcpyHostToDevice));
//     CHECK(hipMemcpy(d_fractionPart, fractionPart, sizeof(double), hipMemcpyHostToDevice));
//     CHECK(hipMemcpy(d_start_time, start_time, sizeof(double), hipMemcpyHostToDevice));
//     // printf("[GPU]GPU数据拷贝完成\n");
//     CHECK(hipEventRecord(data_stop));
//     CHECK(hipEventSynchronize(data_stop));
    
//     //设置使用编号为0的GPU
//     CHECK(hipSetDevice(0));
//     // //设置线程数量
//     int threadPerBlock= 256;
//     int blocksPerGrid = 70;
//     dim3 block(threadPerBlock);
//     //设置块数量
//     dim3 grid(blocksPerGrid);//blocksPerGrid
//     //调用核函数
//     // printf("[GPU]核函数开始运行\n");
//     CHECK(hipEventRecord(gpu_start));

    
   
//     step_calculate_every_pmt<<<grid, block>>>(d_r,d_pos_x,d_pos_y,d_pos_z,g_pmt_x,g_pmt_y,\
//         g_pmt_z,d_intPart,d_fractionPart,d_start_time,pmt_num,g_data_hit,g_data_npe,seed,\
//         d_hittime_result,d_result_size,1,false);
//     CHECK(hipDeviceSynchronize());
//     // dim3 grid_reduce(evtnum/threadPerBlock+1);
//     // evt_calculate_add<<<grid_reduce,block>>>(d_evt_res_list,d_evt_res_back,evtnum,pmt_num);
//     // CHECK(hipDeviceSynchronize());
//     CudaCheckError();
//     CHECK(hipEventRecord(gpu_stop));
//     CHECK(hipEventSynchronize(gpu_stop));
    
//     // printf("[GPU]核函数运行完成\n");
//     // CHECK(hipDeviceSynchronize());
//     CHECK(hipMemcpy(hittime_addr, d_hittime_result, pmt_num*pmt_mem*sizeof(double), hipMemcpyDeviceToHost));
//     // CHECK(hipMemcpy(pmtid_addr, d_pmtid_result, pmt_num*pmt_mem*sizeof(int), hipMemcpyDeviceToHost));
//     CHECK(hipMemcpy(result_size, d_result_size, pmt_num*sizeof(int), hipMemcpyDeviceToHost));

//     // printf("[GPU]GPU运行完成\n");

//     CHECK(hipEventRecord(stop));
//     CHECK(hipEventSynchronize(stop));
//     float total_time,gputime,datatime;
//     //计算用时，精度0.5us
//     CHECK(hipEventElapsedTime(&datatime, data_start, data_stop));
//     CHECK(hipEventElapsedTime(&gputime, gpu_start, gpu_stop));
//     CHECK(hipEventElapsedTime(&total_time, start, stop));
//     CHECK(hipEventDestroy(gpu_start));
//     CHECK(hipEventDestroy(gpu_stop));
//     CHECK(hipEventDestroy(data_start));
//     CHECK(hipEventDestroy(data_stop));
//     CHECK(hipEventDestroy(start));
//     CHECK(hipEventDestroy(stop));        
//     printf("total use time %f ms\n", total_time);
//     printf("gpu use time %f ms\n",gputime);
//     printf("data use time %f ms\n",datatime);
//     printf("data transport back use time %f ms\n",total_time - datatime - gputime);      
//     return total_time;
//     // return 0.0;
// }

// void GPU_exit()
// {
//     //释放GPU内存
//     CHECK(hipFree(d_r));
//     CHECK(hipFree(d_pos_x));
//     CHECK(hipFree(d_pos_y));
//     CHECK(hipFree(d_pos_z));
//     CHECK(hipFree(d_intPart));
//     CHECK(hipFree(d_fractionPart));
//     CHECK(hipFree(d_start_time));
//     CHECK(hipFree(g_pmt_x));
//     CHECK(hipFree(g_pmt_y));
//     CHECK(hipFree(g_pmt_z));
//     CHECK(hipFree(g_data_hit));
//     CHECK(hipFree(g_data_npe));
//     CHECK(hipFree(d_hittime_result));
//     // CHECK(hipFree(d_pmtid_result));
//     CHECK(hipFree(d_result_size));
//     CHECK(hipFree(d_npe_result));
//     CHECK(hipDeviceReset());
// }


// float GPU_Sampling_merge(double *r,double *pos_x,double *pos_y,double *pos_z, \
//     double *intPart, double *fractionPart,double *start_time,\
//     int seed,int step_size,double *hittime_addr,int *result_size)
// {    
//     CHECK(hipMemset(d_hittime_result,0,pmt_num*pmt_mem*sizeof(double)));
    
//     //将CPU内存拷贝到GPU
//     CHECK(hipMemcpy(d_r, r, step_size, hipMemcpyHostToDevice));
//     CHECK(hipMemcpy(d_pos_x, pos_x, step_size, hipMemcpyHostToDevice));
//     CHECK(hipMemcpy(d_pos_y, pos_y, step_size, hipMemcpyHostToDevice));
//     CHECK(hipMemcpy(d_pos_z, pos_z, step_size, hipMemcpyHostToDevice));
//     CHECK(hipMemcpy(d_intPart, intPart, step_size, hipMemcpyHostToDevice));
//     CHECK(hipMemcpy(d_fractionPart, fractionPart, step_size, hipMemcpyHostToDevice));
//     CHECK(hipMemcpy(d_start_time, start_time, step_size, hipMemcpyHostToDevice));
    
//     //设置使用编号为0的GPU
//     CHECK(hipSetDevice(0));
//     // //设置线程数量
//     int threadPerBlock= 256;
//     int blocksPerGrid = 70;
//     dim3 block(threadPerBlock);
//     //设置块数量
//     dim3 grid(blocksPerGrid);//blocksPerGrid  
   
//     step_calculate_every_pmt<<<grid, block>>>(d_r,d_pos_x,d_pos_y,d_pos_z,g_pmt_x,g_pmt_y,\
//         g_pmt_z,d_intPart,d_fractionPart,d_start_time,pmt_num,g_data_hit,g_data_npe,seed,\
//         d_hittime_result,d_result_size,1,true);
//     CHECK(hipDeviceSynchronize());
//     CudaCheckError();
    
//     CHECK(hipMemcpy(hittime_addr, d_hittime_result, pmt_num*pmt_mem*sizeof(double), hipMemcpyDeviceToHost));
//     return 0.0;
// }

// void GPU_Sampling_merge_multi(double *r,double *pos_x,double *pos_y,double *pos_z, \
//     double *intPart, double *fractionPart,double *start_time,\
//     int seed,int step_size,double *hittime_addr,int *hittime_npe)
// {
//     CHECK(hipMalloc((double**)&d_r,step_size*sizeof(double)));
//     CHECK(hipMalloc((double**)&d_pos_x,step_size*sizeof(double)));
//     CHECK(hipMalloc((double**)&d_pos_y,step_size*sizeof(double)));
//     CHECK(hipMalloc((double**)&d_pos_z,step_size*sizeof(double)));
//     CHECK(hipMalloc((double**)&d_intPart,step_size*sizeof(double)));
//     CHECK(hipMalloc((double**)&d_fractionPart,step_size*sizeof(double)));
//     CHECK(hipMalloc((double**)&d_start_time,step_size*sizeof(double)));

//     CHECK(hipMemset(d_hittime_result,0,pmt_num*pmt_mem*sizeof(double)));
//     CHECK(hipMemset(d_npe_result,0,pmt_num*pmt_mem*sizeof(int)));
    
//     //将CPU内存拷贝到GPU
//     CHECK(hipMemcpy(d_r, r, step_size*sizeof(double), hipMemcpyHostToDevice));
//     CHECK(hipMemcpy(d_pos_x, pos_x, step_size*sizeof(double), hipMemcpyHostToDevice));
//     CHECK(hipMemcpy(d_pos_y, pos_y, step_size*sizeof(double), hipMemcpyHostToDevice));
//     CHECK(hipMemcpy(d_pos_z, pos_z, step_size*sizeof(double), hipMemcpyHostToDevice));
//     CHECK(hipMemcpy(d_intPart, intPart, step_size*sizeof(int), hipMemcpyHostToDevice));
//     CHECK(hipMemcpy(d_fractionPart, fractionPart, step_size*sizeof(double), hipMemcpyHostToDevice));
//     CHECK(hipMemcpy(d_start_time, start_time, step_size*sizeof(double), hipMemcpyHostToDevice));
//     //设置使用编号为0的GPU
//     CHECK(hipSetDevice(0));
//     // //设置线程数量
//     int threadPerBlock= 256;
//     int blocksPerGrid = 70;
//     dim3 block(threadPerBlock);
//     //设置块数量
//     dim3 grid(blocksPerGrid);//blocksPerGrid  
   
//     step_calculate_every_pmt<<<grid, block>>>(d_r,d_pos_x,d_pos_y,d_pos_z,g_pmt_x,g_pmt_y,\
//         g_pmt_z,d_intPart,d_fractionPart,d_start_time,pmt_num,g_data_hit,g_data_npe,seed,\
//         d_hittime_result,d_npe_result,step_size,true); 
//     CHECK(hipDeviceSynchronize());
//     CudaCheckError(); 
//     CHECK(hipMemcpy(hittime_addr, d_hittime_result, pmt_num*pmt_mem*sizeof(double), hipMemcpyDeviceToHost)); 
//     CHECK(hipMemcpy(hittime_npe, d_npe_result, pmt_num*pmt_mem*sizeof(int), hipMemcpyDeviceToHost)); 

// }